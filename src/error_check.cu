#include <stdio.h>
#include "common.cuh"


int main() {
    SetGpu();
    float *host_a;
    host_a = new float(4);
    memset(host_a, 0, sizeof(float));

    float *device_a;
    hipError_t error = ErrorCheck(hipMalloc((float **)&device_a, 4), __FILE__, __LINE__);
    hipMemset(device_a , 0, sizeof(float));

    // 从主机到设备拷贝。但是使用的是hipMemcpyDeviceToHost，将会报错
    ErrorCheck(hipMemcpy(device_a, host_a, sizeof(float), hipMemcpyDeviceToHost), __FILE__, __LINE__);

    delete host_a;
    ErrorCheck(hipFree(device_a), __FILE__, __LINE__);
    ErrorCheck(hipDeviceReset(), __FILE__, __LINE__);



    return 0;
}